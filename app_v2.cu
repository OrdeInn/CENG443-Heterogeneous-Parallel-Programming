
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <limits.h>
#include <stdbool.h>

// Number of vertices in the graph
#define MAX 9
#define INF 99999
#define THREADS_BLOCK 1


__global__ void findClosestVertice(int* distance, int* visited, int* global_closest, int num_vertices) {
    
    int dist = INF + 1;
    int vertice = -1;
    int i;

    for (i = 0; i < num_vertices; i++) {
        
        if ((distance[i] < dist) && (visited[i] != 1)) {
            
            dist = distance[i];
            vertice = i;
        }
    }

    global_closest[0] = vertice;
    visited[vertice] = 1;
}

__global__ void relaxEdges(int* graph, int* distance, int* parent_node, int* visited, int* global_closest) {

    int next = blockIdx.x * blockDim.x + threadIdx.x;
    int source = global_closest[0];

    int edge = graph[source * MAX + next];
    int new_dist = distance[source] + edge;

    if ((edge != 0) && (visited[next] != 1) && (new_dist < distance[next])) {

        distance[next] = new_dist;
        parent_node[next] = source;
    }

}



int main(){

	int n = 9;
	int start = 0;

	int* graph = (int*) malloc(MAX * MAX * sizeof(int));
	int* distance = (int*) malloc(MAX * sizeof(int));
	int* visited = (int*) malloc(MAX * sizeof(int));
    int* parent_vertice = (int*) malloc(MAX * sizeof(int));
	
    int Graph[9][9] = { { 0, 4, 0, 0, 0, 0, 0, 8, 0 },
                        { 4, 0, 8, 0, 0, 0, 0, 11, 0 },
                        { 0, 8, 0, 7, 0, 4, 0, 0, 2 },
                        { 0, 0, 7, 0, 9, 14, 0, 0, 0 },
                        { 0, 0, 0, 9, 0, 10, 0, 0, 0 },
                        { 0, 0, 4, 14, 10, 0, 2, 0, 0 },
                        { 0, 0, 0, 0, 0, 2, 0, 1, 6 },
                        { 8, 11, 0, 0, 0, 0, 1, 0, 7 },
                        { 0, 0, 2, 0, 0, 0, 6, 7, 0 } }; 


	for(int j=0 ; j < n; j++){

		for(int i = 0; i < n; i++){

			graph[i + (j*n)] = Graph[j][i];
		}
	}

	for (int i = 0; i < n; i++) {

        if(graph[i + (start*n)] == 0){
    
    		distance[i] = INF; 
        }
        else{

    		distance[i] = graph[i + (start*n)]; 
        }

		visited[i] = 0;
	}

    for(int i = 0; i < n; i++){

        parent_vertice[i] = -1;
    }

    distance[start] = 0;

	hipError_t err = hipSuccess;

	int size = MAX  * sizeof(int);

	int *d_graph = NULL;
    err = hipMalloc((void **)&d_graph, MAX * MAX * sizeof(int));

	if (err != hipSuccess){

       	fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
       	exit(EXIT_FAILURE);
    }

	int *d_distance = NULL;
    err = hipMalloc((void **)&d_distance, size);

	if (err != hipSuccess){

       	fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
       	exit(EXIT_FAILURE);
    }

	int *d_visited = NULL;
    err = hipMalloc((void **)&d_visited, size);

	if (err != hipSuccess){

       	fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
       	exit(EXIT_FAILURE);
    }

    int *d_closest = NULL;
    err = hipMalloc((void **)&d_closest, sizeof(int));

	if (err != hipSuccess){

       	fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
       	exit(EXIT_FAILURE);
    }

    int *d_parent_vertice = NULL;
    err = hipMalloc((void **)&d_parent_vertice, size);

	if (err != hipSuccess){

       	fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
       	exit(EXIT_FAILURE);
    }

	err = hipMemcpy(d_graph, graph, MAX * MAX * sizeof(int), hipMemcpyHostToDevice);
	err = hipMemcpy(d_distance, distance, size, hipMemcpyHostToDevice);
	err = hipMemcpy(d_visited, visited, size, hipMemcpyHostToDevice);
	err = hipMemcpy(d_parent_vertice, parent_vertice, size, hipMemcpyHostToDevice);

	if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

	dim3 threads(1, 1);
    dim3 blocks(1, 1);

    dim3 gridRelax(MAX / THREADS_BLOCK, 1);
    dim3 blockRelax(THREADS_BLOCK, 1);


    //Measure execution Time
    hipEvent_t e_start, e_stop;
    hipEventCreate(&e_start);
    hipEventCreate(&e_stop);

    hipEventRecord(e_start);


    for(int i = 0; i < n; i++){

        findClosestVertice<<<blocks, threads>>>(d_distance, d_visited, d_closest, n);
        relaxEdges<<<gridRelax, blockRelax>>>(d_graph, d_distance, d_parent_vertice, d_visited, d_closest);
    }

    hipEventRecord(e_stop);

    err = hipGetLastError();

	if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch gpu kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

	// Copy the device result vector in device memory to the host result vector
    // in host memory.
    err = hipMemcpy(distance, d_distance, size, hipMemcpyDeviceToHost);
    err = hipMemcpy(parent_vertice, d_parent_vertice, size, hipMemcpyDeviceToHost);


    // Print execution time
    hipEventSynchronize(e_stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, e_start, e_stop);

    printf("\napp_v2 executed in %f milliseconds", milliseconds);

	// Printing the distance
 	printf("\nVertex \t Distance from Source\n");
    for (int i = 0; i < n; i++){
        printf("%d\t\t%d\n", i, distance[i]);
	}
    	
	//Free gpu memories
    hipFree(d_graph);
    hipFree(d_distance);
    hipFree(d_parent_vertice);
    hipFree(d_visited);

    //Free cpu memories
    free(distance);
    free(parent_vertice);
    free(visited);
    free(graph);

    return EXIT_SUCCESS;
}